
#include <hip/hip_runtime.h>
/// CUDA API functions with a nice interface for Rust

extern "C" {
void* numba_Allocate(size_t s) {
  void* ptr;
  hipMalloc(&ptr, s);
  return ptr;
}

void numba_Free(void* ptr) {
  hipFree(ptr);
}

void numba_CopyToDevice(void* devDst, void* hostSrc, size_t s) {
  hipMemcpy(devDst, hostSrc, s, hipMemcpyHostToDevice);
}

void numba_CopyToHost(void* hostDst, void* devSrc, size_t s) {
  hipMemcpy(hostDst, devSrc, s, hipMemcpyDeviceToHost);
}

void numba_CopyDeviceToDevice(void* devDst, void* devSrc, size_t s) {
  hipMemcpy(devDst, devSrc, s, hipMemcpyDeviceToDevice);
}

void numba_Memset(void* devPtr, int value, size_t s) {
  hipMemset(devPtr, value, s);
}
}