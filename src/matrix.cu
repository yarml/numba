
#include <hip/hip_runtime.h>
static __global__ void
kernel_linearAdd(float* a, float* b, float* c, size_t N) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < N) {
    c[index] = a[index] + b[index];
  }
}

// Matrix multiplication kernel:
// Path: src/matrix.cu
static __global__ void
kernel_matrixMul(float* a, float* b, float* c, size_t N, size_t M, size_t R) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < N && col < R) {
    float sum = 0;
    for (int i = 0; i < M; i++) {
      sum += a[row * M + i] * b[i * R + col];
    }
    c[row * R + col] = sum;
  }
}

extern "C" {
void linear_add(float* a, float* b, float* c, size_t N) {
  size_t size = N * sizeof(float);
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  size_t threadsPerBlock = 256;
  size_t blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  kernel_linearAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

void matrix_mul(float* a, float* b, float* c, size_t N, size_t M, size_t R) {
  size_t sizeA = N * M * sizeof(float);
  size_t sizeB = M * R * sizeof(float);
  size_t sizeC = N * R * sizeof(float);
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, sizeA);
  hipMalloc(&d_b, sizeB);
  hipMalloc(&d_c, sizeC);
  hipMemcpy(d_a, a, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeB, hipMemcpyHostToDevice);
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid(
    (R + threadsPerBlock.x - 1) / threadsPerBlock.x,
    (N + threadsPerBlock.y - 1) / threadsPerBlock.y
  );
  kernel_matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N, M, R);
  hipMemcpy(c, d_c, sizeC, hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
}
